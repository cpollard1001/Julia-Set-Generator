#include "hip/hip_runtime.h"
#include "png.cpp"
#include <complex>
#include <string.h>
#include <limits>
#include <cstdlib>
#include <iostream>
void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)
void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}
void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

using namespace std;

double sample(double x, double y, complex<double> comp) {
  double mag = 0;
  int count = 0;
  complex<double> dz(1, 0);
  complex<double> point(x,y);
  complex<double> two(2,0);
  while (count < 1024 && mag < 1e20) {
    dz = two*dz*point;
    point = point*point + comp;
    mag = abs(point);
    count++;
  }
  double dist = log(mag) * mag / abs(dz);
  return dist;
}

void juliaSet(double buffer[], int width, int height,double xCenter, double yCenter,double gridWidth,int samplesPerPixel,complex<double> comp){
  for(int i = 0; i < width; i++){
    for(int j = 0; j < height; j++){
      double intensity = 0;
      for(int k = 0; k < samplesPerPixel; k++){
        double randNum = ((double)rand()/(double)RAND_MAX);
        double x = xCenter + gridWidth * (i - width / 2) / width + randNum / width;
        randNum = ((double)rand()/(double)RAND_MAX);
        double y = yCenter + gridWidth * height / width * (j - height / 2) / height + randNum / height;
        intensity += sample(x,y,comp);
      }
      intensity = intensity / samplesPerPixel;
      buffer[i*width+j] = intensity;
    }
    if(i%50==0) printf("%d\n",i);
  }
}

void getFileName(char* fileName,char* folder,int frame,char* subFolder){
  char temp[20];
  strcpy(fileName,folder);
  strcat(fileName,"/");
  strcat(fileName,subFolder);
  strcat(fileName,"/");
  sprintf(temp, "%05d", frame);
  strcat(fileName,temp);
  strcat(fileName,".png");
}

void writeImages(double* buffer,int width,int height,int frame,char* folder){
  renderSetting setting;
  char fileName[30];

  char subFolder[20] = "Red";
  getFileName(fileName, folder, frame, subFolder);

  printf(fileName);

  /*setting.map = LOG;
  setting.cs = RED;
  setting.add = 15;//16;
  setting.mult = 17.0/255;//19.7/255;
  writeImage(fileName, width, height, buffer, &setting);*/

  strcpy(subFolder,"Blue");
  getFileName(fileName, folder, frame, subFolder);
  setting.map = LOG;
  setting.cs = BLUE;
  setting.add = 15;//16;
  setting.mult = 17.0/255;//19.7/255;
  writeImage(fileName, width, height, buffer, &setting);

  strcpy(subFolder,"Bump");
  getFileName(fileName, folder, frame, subFolder);
  setting.map = LOG;
  setting.cs = BW;
  setting.add = 15;//16;
  setting.mult = 17.0/255;//19.7/255;
  writeImage(fileName, width, height, buffer, &setting);

  strcpy(subFolder,"Disp");
  getFileName(fileName, folder, frame, subFolder);
  setting.cs = INV;
  setting.map = EXP;
  setting.exp = .9;
  setting.add = 0;
  setting.mult = 6000.0/255;
  writeImage(fileName, width, height, buffer, &setting);
}

__device__ double cudasample(double x, double y, double compR, double compI) {
  double mag = 0;
  int count = 0;
  double dzR = 1;
  double dzI = 0;
  double pointR = x;
  double pointI = y;
  while (count < 1024 && mag < 1e5) {
    double tmpdzR = 2 * (dzR * pointR - dzI * pointI);
    dzI = 2 * (dzR*pointI + dzI*pointR);
    dzR = tmpdzR;

    double tmpPointR = pointR*pointR - pointI*pointI + compR;
    pointI = 2 * pointR *pointI + compI;
    pointR = tmpPointR;

    mag = sqrt(pointR*pointR + pointI*pointI);
    count++;
  }
  double dist = logf(mag) / sqrt(dzR*dzR+dzI*dzI) * mag;
  return dist;
}

__global__ void cudaJulia(double buffer[], int width, int height,double xCenter, double yCenter,double gridWidth,int samplesPerPixel,double compR, double compI,int rep) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + rep*4096*blockDim.x;
  if(index < width * height){
    int i = index % width;
    int j = index / height;
    double intensity = 0;
    for(int k = 0; k < samplesPerPixel; k++){
      double random = ((double)k)/samplesPerPixel/width;
      double x = xCenter + gridWidth * (i - width / 2) / width + random / width;
      random = ((index+k)%samplesPerPixel)/samplesPerPixel/height;
      double y = yCenter + gridWidth * height / width * (j - height / 2) / height + random / height;
      intensity += cudasample(x,y,compR, compI);
    }
    intensity = intensity / samplesPerPixel;
    buffer[index] = intensity;
  }
}

int main(int argc, char *argv[])
{
	int width = 1024;
	int height = 1024;
  double xCenter = 0;//.377;
  double yCenter = 0;//-.343;
  double gridWidth = 1.0;
  double samplesPerPixel = 1;
  char folder[20] = "Test";

  int t = 0;
  double animCenterX = -.77;//-.433;
  double animCenterY = .156;//.62;
  double animRadX = .025;
  double animRadY = .01;//.03;
  double speed = .006;

  int blocksize = 1024;
  int gridsize = width*height/blocksize;
  int numReps = 1;
  if(gridsize > 4096){
    numReps = gridsize/4096;
    gridsize = 4096;
  }

  int numFrames = 2 * 3.1415926535 / speed;
  printf("%d frames total\n",numFrames);


  int bufferSize = width * height * sizeof(double);
  double* buffer = (double *) malloc(bufferSize);
  double* bufferd;
  cudaCheck(hipMalloc( (void**)&bufferd, bufferSize ));

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( gridsize, 1 );

  while(t < numFrames){
    double xJParam = animCenterX + animRadX * cos(t*speed);
    double yJParam = animCenterY + animRadY * sin(t*speed);
    complex<double> comp(xJParam,yJParam);

    printf("Frame %d, r = %f, i = %f\n",t,xJParam,yJParam);
    //call cudaJulia, pass all the same arguments
    int rep = 0;
    while(rep<numReps){
      cudaJulia<<<dimGrid, dimBlock>>>(bufferd, width, height, xCenter, yCenter, gridWidth, samplesPerPixel, comp.real(), comp.imag(),rep);
      cudaCheckLastError("Failure");
      rep++;
    }
    //copy memory back to cpu
    hipMemcpy( buffer, bufferd, bufferSize, hipMemcpyDeviceToHost );
    //juliaSet(buffer, width, height, xCenter, yCenter, gridWidth, samplesPerPixel, comp);
    //sort / analyze histogram if needed, hopefully not though
    //write images
    writeImages(buffer,width,height,t,folder);

    t = t + 1;
  }

  hipFree( bufferd );
	free(buffer);

	return 0;
}
